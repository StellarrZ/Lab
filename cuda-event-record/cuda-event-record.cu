#include <iostream>
#include <vector>
#include <mutex>
#include <condition_variable>

#include "hip/hip_runtime.h"

#include <cstdio>
#include <cstdlib>

#define CUDACHECK(cmd) do {                         \
  hipError_t e = cmd;                              \
  if( e != hipSuccess ) {                          \
    printf("Failed: Cuda error %s:%d '%s'\n",       \
        __FILE__,__LINE__,hipGetErrorString(e));   \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)


// empty kernel
__global__ void NoOpKernel() {}


// for blocking stream to wait for host signal
class Event {
 private:
  std::mutex mtx_condition_;
  std::condition_variable condition_;
  bool signalled = false;

 public:
  void Signal() {
    {
      std::lock_guard<decltype(mtx_condition_)> lock(mtx_condition_);
      signalled = true;
    }
    condition_.notify_all();
  }

  void Wait() {
    std::unique_lock<decltype(mtx_condition_)> lock(mtx_condition_);
    while (!signalled) {
      condition_.wait(lock);
    }
  }
};

void CUDART_CB block_op_host_fn(void* arg) {
  Event* evt = (Event*)arg;
  evt->Wait();
}


static hipEvent_t end_event;

int main() {
  hipStream_t stream1, stream2;
  CUDACHECK(hipStreamCreate(&stream1));
  CUDACHECK(hipStreamCreate(&stream2));

  int num_events = 150; // hangs if THIS >= 57
  std::vector<std::shared_ptr<Event>> event_vec;

  for (int i = 0; i < num_events; i++) {
    NoOpKernel<<<1, 128, 0, stream1>>>();

    event_vec.push_back(std::make_shared<Event>());
    hipLaunchHostFunc(stream1, block_op_host_fn, event_vec.back().get());

    std::cout << "Before recording #" << i << std::endl;
    CUDACHECK(hipEventCreate(&end_event));
    CUDACHECK(hipEventRecord(end_event, stream1));
    // CUDACHECK(cudaEventRecord(end_event, stream2));
    std::cout << "After recording  #" << i << std::endl;

    // NoOpKernel<<<1, 128, 0, stream1>>>();
  }

  for (int i = 0; i < num_events; i++) {
    event_vec[i]->Signal();
  }

  // clean up
  CUDACHECK(hipDeviceSynchronize());
  CUDACHECK(hipEventDestroy(end_event));
  CUDACHECK(hipStreamDestroy(stream1));
  CUDACHECK(hipStreamDestroy(stream2));
  return 0;
}