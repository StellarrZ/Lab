#include <iostream>
#include <vector>
#include <mutex>
#include <condition_variable>
#include <thread>

#include "hip/hip_runtime.h"

#include <cstdio>
#include <cstdlib>

#define CUDACHECK(cmd) do {                         \
  hipError_t e = cmd;                              \
  if( e != hipSuccess ) {                          \
    printf("Failed: Cuda error %s:%d '%s'\n",       \
        __FILE__,__LINE__,hipGetErrorString(e));   \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)


// empty kernel
__global__ void NoOpKernel() {}


// for blocking stream to wait for host signal
class Event {
 private:
  std::mutex mtx_condition_;
  std::condition_variable condition_;
  bool signalled = false;

 public:
  void Signal() {
    {
      std::lock_guard<decltype(mtx_condition_)> lock(mtx_condition_);
      signalled = true;
    }
    condition_.notify_all();
  }

  void Wait() {
    std::unique_lock<decltype(mtx_condition_)> lock(mtx_condition_);
    while (!signalled) {
      condition_.wait(lock);
    }
  }
};

void CUDART_CB block_op_host_fn(void* arg) {
  Event* evt = (Event*)arg;
  evt->Wait();
}


static hipEvent_t end_event;

// check if other threads are non-blocking
static void checkOtherThread() {
  std::this_thread::sleep_for(std::chrono::milliseconds(2000));
  for (int i = 0; i < 10; i++) {
    std::cout << "Non-blocking" << i << std::endl;
  }
}

// keep launching empty kernels
static void launchKernels(hipStream_t stream) {
  for (int i = 0; i < 1000; i++) {
    NoOpKernel<<<1, 128, 0, stream>>>();
    std::cout << "Done KernelLaunch #" << i << std::endl;
  }
}

static void blocker(hipStream_t stream) {
  int num_events = 150; // hangs if THIS >= 57
  std::vector<std::shared_ptr<Event>> event_vec;

  for (int i = 0; i < num_events; i++) {
    event_vec.push_back(std::make_shared<Event>());
    hipLaunchHostFunc(stream, block_op_host_fn, event_vec.back().get());

    std::cout << "Before recording #" << i << std::endl;
    CUDACHECK(hipEventCreate(&end_event));
    CUDACHECK(hipEventRecord(end_event, stream));
    std::cout << "After recording  #" << i << std::endl;
  }

  for (int i = 0; i < num_events; i++) {
    event_vec[i]->Signal();
  }
}


int main() {
  hipStream_t stream1, stream2;
  CUDACHECK(hipStreamCreate(&stream1));
  CUDACHECK(hipStreamCreate(&stream2));

  auto t1 = std::thread(blocker, stream1);
  auto t2 = std::thread(launchKernels, stream2);
  auto t3 = std::thread(checkOtherThread);

  t1.join();
  t2.join();
  t3.join();

  // clean up
  CUDACHECK(hipDeviceSynchronize());
  CUDACHECK(hipEventDestroy(end_event));
  CUDACHECK(hipStreamDestroy(stream1));
  CUDACHECK(hipStreamDestroy(stream2));
  return 0;
}