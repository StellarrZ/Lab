#include <iostream>
#include <hip/hip_runtime.h>


#define CUDACHECK(cmd) do {                         \
  hipError_t e = cmd;                              \
  if( e != hipSuccess ) {                          \
    printf("Failed: Cuda error %s:%d '%s'\n",       \
        __FILE__,__LINE__,hipGetErrorString(e));   \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)


/* Kernel 1 - Bad data locality */
__global__ void memcpyD2D_a(float* src, float* dst, int num_elem) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;  // 0 + threadIdx.x
  int stride = blockDim.x * gridDim.x;  // blockDim.x * 1
  int tile = num_elem / stride;

  for (int k = 0; k < tile; k++) {
    dst[k + tid * tile] = src[k + tid * tile];
  }
}

/* Kernel 2 - Okay data locality */
__global__ void memcpyD2D_b(float* src, float* dst, int num_elem) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;  // 0 + threadIdx.x
  int stride = blockDim.x * gridDim.x;  // blockDim.x * 1

  for (int k = 0; k < num_elem; k += stride) {
    dst[k + tid] = src[k + tid];
  }
}

#define FETCH_FLOAT4(pointer) (reinterpret_cast<float4*>(&(pointer))[0])
/* Kernel 3 - Vectorized mem access */
__global__ void memcpyD2D_c(float* src, float* dst, int num_elem) {
  int tid = (blockIdx.x * blockDim.x + threadIdx.x) * 4;  // (0 + threadIdx.x) * 4
  int stride = blockDim.x * gridDim.x * 4;  // blockDim.x * 1 * 4

  for (int k = 0; k < num_elem; k += stride) {
    float4 src4 = FETCH_FLOAT4(src[k + tid]);
    FETCH_FLOAT4(dst[k + tid]) = src4;
  }
}


int main(int argc, char* argv[]) {
  CUDACHECK(hipSetDevice(0));

  hipStream_t stream;
  CUDACHECK(hipStreamCreate(&stream));

  const size_t num_of_bytes = 4 << 20;
  float* h_buf;
  float* d_src;
  float* d_dst;

  h_buf = (float*)malloc(num_of_bytes);
  CUDACHECK(hipMalloc(&d_src, num_of_bytes));
  CUDACHECK(hipMalloc(&d_dst, num_of_bytes));

  int num_elem = num_of_bytes / sizeof(float);

  for (int i = 0; i < num_elem; i++) {
    // h_buf[i] = static_cast<float>(rand()) / static_cast<float>(RAND_MAX);
    h_buf[i] = 1.0;
  }
  CUDACHECK(hipMemcpy(d_src, h_buf, num_of_bytes, hipMemcpyHostToDevice));

  hipEvent_t start, stop;
  CUDACHECK(hipEventCreate(&start));
  CUDACHECK(hipEventCreate(&stop));

  for (int j = 0; j < 100; j++) {
    CUDACHECK(hipEventRecord(start, stream));
    // memcpyD2D_a<<<1, 1024, 0, stream>>>(d_src, d_dst, num_elem);
    // memcpyD2D_b<<<1, 1024, 0, stream>>>(d_src, d_dst, num_elem);
    memcpyD2D_c<<<1, 1024, 0, stream>>>(d_src, d_dst, num_elem);
    CUDACHECK(hipEventRecord(stop, stream));

    // if (cudaGetLastError() != cudaSuccess) {
    //   std::cout << "Got ERROR" << std::endl;
    // }

    CUDACHECK(hipEventSynchronize(stop));
    float milliseconds = 0;
    CUDACHECK(hipEventElapsedTime(&milliseconds, start, stop));

    std::cout<< "Took time(ms) : " << milliseconds << std::endl;

    CUDACHECK(hipStreamSynchronize(stream));
  }


  size_t cnt = 0;
  float* check = (float*)malloc(num_of_bytes);
  CUDACHECK(hipMemcpy(check, d_dst, num_of_bytes, hipMemcpyDeviceToHost));
  for (int i = 0; i < num_elem; i++) {
    if (check[i] < 0.9 || check[i] > 1.1) {
      cnt++;
      // printf("Not match: %d  %f\n", i, check[i]);
    }
  }
  std::cout << cnt << std::endl;
}