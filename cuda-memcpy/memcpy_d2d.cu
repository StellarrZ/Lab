#include<cstdio>
#include<iostream>
#include<hip/hip_runtime.h>
#include<chrono>


int main() {
  hipSetDevice(0);

  size_t num_bytes = 25 << 20;
  float *src, *dst;
  hipMalloc(&src, num_bytes);
  hipMalloc(&dst, num_bytes);

  for (int i = 0; i < 100; i++) {
    auto start = std::chrono::system_clock::now();
    hipMemcpy(dst, src, num_bytes, hipMemcpyDeviceToDevice);
    auto end = std::chrono::system_clock::now();
    std::cout << std::chrono::duration_cast<std::chrono::nanoseconds>(
                    end.time_since_epoch() - start.time_since_epoch()).count()
              << " ns" 
              << std::endl;
  }
}