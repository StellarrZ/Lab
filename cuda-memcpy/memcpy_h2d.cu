#include <iostream>
#include <chrono>
#include <hip/hip_runtime.h>


int main() {
  hipSetDevice(0);
  float* host = (float*)malloc(32);
  float* device;
  hipMalloc(&device, 32);
  
  hipEvent_t sta, end;
  hipEventCreate(&sta);
  hipEventCreate(&end);

  for (int i = 0; i < 100; i++) {
    // cudaEventRecord(sta);
    auto begin = std::chrono::steady_clock::now();
    hipMemcpy(device, host, 32, hipMemcpyHostToDevice);
    auto final = std::chrono::steady_clock::now();
    // cudaEventRecord(end);

    // float ms;
    // cudaEventSynchronize(end);
    // cudaEventElapsedTime(&ms, sta, end);

    std::cout << (float)std::chrono::duration_cast<std::chrono::nanoseconds>(final - begin).count() / 1E6 << std::endl;
    // std::cout << ms << "  " << (float)std::chrono::duration_cast<std::chrono::nanoseconds>(final - begin).count() / 1E6 << std::endl;
  }

  return 0;
}