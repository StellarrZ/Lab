#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cooperative_groups/memcpy_async.h>
#include <cuda/pipeline>


__device__ void compute(int* global_out, int const* shared_in) {

}

__global__ void with_staging(int* global_out, int const* global_in, size_t size, size_t batch_sz) {
    auto grid = cooperative_groups::this_grid();
    auto block = cooperative_groups::this_thread_block();

    // printf("%lld  %d  %d  %d  %d\n", grid.size(), block.size(), blockIdx.x, blockDim.x, threadIdx.x);
    // printf("%d\n", block.group_index().x);
    assert(size == batch_sz * grid.size()); // Assume input size fits batch_sz * grid_size

    constexpr size_t stages_count = 2; // Pipeline with two stages
    // Two batches must fit in shared memory:
    extern __shared__ int shared[];  // stages_count * block.size() * sizeof(int) bytes
    size_t shared_offset[stages_count] = { 0, block.size() }; // Offsets to each batch

    // Allocate shared storage for a two-stage cuda::pipeline:
    __shared__ cuda::pipeline_shared_state<
        cuda::thread_scope::thread_scope_block,
        stages_count
    > shared_state;
    auto pipeline = cuda::make_pipeline(block, &shared_state);

    // Each thread processes `batch_sz` elements.
    // Compute offset of the batch `batch` of this thread block in global memory:
    auto block_batch = [&](size_t batch) -> int {
      return block.group_index().x * block.size() + grid.size() * batch;
    };

    // Initialize first pipeline stage by submitting a `memcpy_async` to fetch a whole batch for the block:
    if (batch_sz == 0) return;
    pipeline.producer_acquire();
    cuda::memcpy_async(block, shared + shared_offset[0], global_in + block_batch(0), sizeof(int) * block.size(), pipeline);
    pipeline.producer_commit();

    // Pipelined copy/compute:
    for (size_t batch = 1; batch < batch_sz; ++batch) {
        // Stage indices for the compute and copy stages:
        size_t compute_stage_idx = (batch - 1) % 2;
        size_t copy_stage_idx = batch % 2;

        size_t global_idx = block_batch(batch);

        // Collectively acquire the pipeline head stage from all producer threads:
        pipeline.producer_acquire();

        // Submit async copies to the pipeline's head stage to be
        // computed in the next loop iteration
        cuda::memcpy_async(block, shared + shared_offset[copy_stage_idx], global_in + global_idx, sizeof(int) * block.size(), pipeline);
        // Collectively commit (advance) the pipeline's head stage
        pipeline.producer_commit();

        // Collectively wait for the operations commited to the
        // previous `compute` stage to complete:
        pipeline.consumer_wait();

        // Computation overlapped with the memcpy_async of the "copy" stage:
        compute(global_out + global_idx, shared + shared_offset[compute_stage_idx]);

        // Collectively release the stage resources
        pipeline.consumer_release();
    }

    // Compute the data fetch by the last iteration
    pipeline.consumer_wait();
    compute(global_out + block_batch(batch_sz-1), shared + shared_offset[(batch_sz - 1) % 2]);
    pipeline.consumer_release();
}


__global__ void dumb(int* buff) {
  printf("%d  %d  %d \n", blockIdx.x, blockDim.x, threadIdx.x);
}


int main() {
  hipStream_t stream;
  hipStreamCreate(&stream);

  int* global_in;
  int* global_out;
  hipMalloc(&global_in, 8 * 4);
  hipMalloc(&global_out, 8 * 4);

  const int num_blocks = 1;
  const int thread_per_block = 256;
  const int batch_sz = 1;
  const int size = num_blocks * thread_per_block * batch_sz;

  with_staging<<<num_blocks, thread_per_block, 1024, stream>>>(
    global_out, global_in, size, batch_sz);
  
  // dumb<<<1, 256, 1024, stream>>>(global_out);
  hipStreamSynchronize(stream);
}