#include <iostream>
#include <thread>
// #include <functional>
#include <future>
#include <algorithm>
#include <vector>
#include <hip/hip_runtime.h>

__global__ void NoOpKernel() {}

using namespace std;

float ThreadFn(hipEvent_t start_evt, vector<hipEvent_t>& evts) {
  float ms = 0.0, ret = -1.0;
  for (auto evt : evts) {
    hipEventSynchronize(evt);
    hipEventElapsedTime(&ms, start_evt, evt);
    ret = max(ret, ms);
    cout << ms << endl;
  }

  return ret;
}

int main() {
  hipStream_t stream;
  hipStreamCreate(&stream);

  vector<hipEvent_t> evts;

  hipEvent_t start_evt;
  hipEventCreate(&start_evt);
  hipEventRecord(start_evt);

  for (int i = 0; i < 20; i++) {
    NoOpKernel<<<1, 128, 0, stream>>>();
    hipEvent_t evt;
    hipEventCreate(&evt);
    hipEventRecord(evt, stream);
    evts.push_back(evt);
  }

  // std::future<float> val_future
  auto val_future = async(launch::async, ThreadFn, start_evt, ref(evts));
  cout << val_future.get() << endl;

  return 0;
}