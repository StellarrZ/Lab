#include <iostream>
#include <thread>
// #include <functional>
#include <future>
#include <algorithm>
#include <vector>
#include <hip/hip_runtime.h>

__global__ void NoOpKernel() {}

using namespace std;

typedef struct packedData {
  __uint128_t data_amount;
  hipEvent_t start_evt;
  std::vector<hipEvent_t> end_evts;
} PackedData;

class C {
public:
  bool b_dumb;
  PackedData data;

  C(): b_dumb(true),
       data({0, nullptr, vector<hipEvent_t>()}) {}
};

float ThreadFn(PackedData* data_ptr) {
  float ms = 0.0, ret = -1.0;
  for (auto evt : data_ptr->end_evts) {
    hipEventSynchronize(evt);
    hipEventElapsedTime(&ms, data_ptr->start_evt, evt);
    ret = max(ret, ms);
    cout << ms << endl;
  }

  return ret;
}

int main() {
  hipStream_t stream;
  hipStreamCreate(&stream);

  C c;

  hipEventCreate(&c.data.start_evt);
  hipEventRecord(c.data.start_evt);

  for (int i = 0; i < 20; i++) {
    NoOpKernel<<<1, 128, 0, stream>>>();
    hipEvent_t evt;
    hipEventCreate(&evt);
    hipEventRecord(evt, stream);
    c.data.end_evts.push_back(evt);
  }

  // std::future<float> val_future
  auto val_future = async(launch::async, ThreadFn, &c.data);
  cout << val_future.get() << endl;

  return 0;
}