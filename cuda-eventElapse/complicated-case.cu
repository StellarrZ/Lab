#include <hip/hip_runtime.h>

#include <thread>
#include <chrono>
#include <cstdio>

__global__ void cuSleep() {
  for (int i = 0; i < 10; i++) {  // 10 ms
    __nanosleep(1000000U);  // 1 ms
  }
}

__global__ void noOp() {}


int main() {
  hipStream_t stream;
  hipStreamCreate(&stream);

  hipEvent_t start_evt_1, end_evt_1;
  hipEvent_t start_evt_2, end_evt_2;
  hipEventCreate(&start_evt_1);
  hipEventCreate(&end_evt_1);
  hipEventCreate(&start_evt_2);
  hipEventCreate(&end_evt_2);

  float ms_1, ms_2;

  for (int i = 0; i < 100; i++) {
    // cudaEventRecord(start_evt_1, stream);
    // cuSleep<<<1, 32, 0, stream>>>();
    // cuSleep<<<1, 32, 0, stream>>>();
    // cuSleep<<<1, 32, 0, stream>>>();
    // cuSleep<<<1, 32, 0, stream>>>();
    // cuSleep<<<1, 32, 0, stream>>>();
    // cudaEventRecord(end_evt_1, stream);

    // cudaEventRecord(start_evt_2, stream);
    // cuSleep<<<1, 32, 0, stream>>>();
    // cuSleep<<<1, 32, 0, stream>>>();
    // cuSleep<<<1, 32, 0, stream>>>();
    // cuSleep<<<1, 32, 0, stream>>>();
    // cuSleep<<<1, 32, 0, stream>>>();
    // cudaEventRecord(end_evt_2, stream);

    hipEventRecord(start_evt_2, stream);
    noOp<<<1, 32, 0, stream>>>();
    noOp<<<1, 32, 0, stream>>>();
    noOp<<<1, 32, 0, stream>>>();
    noOp<<<1, 32, 0, stream>>>();
    noOp<<<1, 32, 0, stream>>>();
    hipEventRecord(end_evt_2, stream);

    hipEventRecord(start_evt_1, stream);
    cuSleep<<<1, 32, 0, stream>>>();
    cuSleep<<<1, 32, 0, stream>>>();
    cuSleep<<<1, 32, 0, stream>>>();
    cuSleep<<<1, 32, 0, stream>>>();
    cuSleep<<<1, 32, 0, stream>>>();
    hipEventRecord(end_evt_1, stream);

    // std::this_thread::sleep_for(std::chrono::milliseconds(200));  // 200 ms

    hipEventSynchronize(end_evt_1);
    hipEventElapsedTime(&ms_1, start_evt_1, end_evt_1);

    hipEventSynchronize(end_evt_2);
    hipEventElapsedTime(&ms_2, start_evt_2, end_evt_2);

    printf("%f ms  %f ms\n", ms_1, ms_2);
  }


  return 0;
}