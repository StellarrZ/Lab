#include <hip/hip_runtime.h>

#include <thread>
#include <chrono>
#include <cstdio>

__global__ void cuSleep() {
  for (int i = 0; i < 10; i++) {  // 10 ms
    __nanosleep(1000000U);  // 1 ms
  }
}


int main() {
  hipStream_t stream;
  hipStreamCreate(&stream);

  hipEvent_t start_evt, end_evt;
  hipEventCreate(&start_evt);
  hipEventCreate(&end_evt);

  float ms;

  for (int i = 0; i < 100; i++) {
    hipEventRecord(start_evt, stream);
    cuSleep<<<1, 32, 0, stream>>>();
    hipEventRecord(end_evt, stream);

    std::this_thread::sleep_for(std::chrono::milliseconds(200)); // 200 ms

    hipEventSynchronize(end_evt);
    hipEventElapsedTime(&ms, start_evt, end_evt);

    printf("%f ms\n", ms);
  }


  return 0;
}