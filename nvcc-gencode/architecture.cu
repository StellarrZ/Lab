#include <hip/hip_runtime.h>

#include <cstdio>


int main() {
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, 0);
  if (hipGetLastError() != hipSuccess) {
    fprintf(stderr, "Failed to get CUDA device #0.\n");
    printf(" ");
  } else {
    int generation = prop.major * 10 + prop.minor;
    printf("NVCC_GENCODE=\"-gencode arch=compute_%d,code=sm_%d\"", generation, generation);
  }

  return 0;
}