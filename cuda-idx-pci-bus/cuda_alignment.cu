#include <cstdio>
#include <hip/hip_runtime.h>


int main() {
  int num_gpu;
  hipGetDeviceCount(&num_gpu);

  hipDeviceProp_t property;

  for (int i = 0; i < num_gpu; i++) {
    hipGetDeviceProperties(&property, i);
    printf("%ld  %ld  %ld\n", 
            property.textureAlignment, 
            property.texturePitchAlignment, 
            property.surfaceAlignment);
  }
  return 0;
}