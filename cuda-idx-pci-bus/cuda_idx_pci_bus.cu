#include <iostream>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <hwloc.h>


int main() {
  int num_gpu;
  hipGetDeviceCount(&num_gpu);

  hipDeviceProp_t property;
  std::cout << "     pciDomainID, pciBusID, pciDeviceID\n" << std::endl;
  for (int i = 0; i < num_gpu; i++) {
    hipGetDeviceProperties(&property, i);
    printf("[%d]  %d, %d, %d\n", i, property.pciDomainID, property.pciBusID, property.pciDeviceID);
  }

  std::cout << std::endl;

  hwloc_topology_t topology;
  hwloc_topology_init(&topology);
  // we need to handle hwloc1.x and hwloc2.x differently
#if HWLOC_API_VERSION < 0x20000
  hwloc_topology_set_flags(topology, HWLOC_TOPOLOGY_FLAG_IO_DEVICES);
#else
  hwloc_topology_set_io_types_filter(topology, HWLOC_TYPE_FILTER_KEEP_ALL);
#endif
  hwloc_topology_load(topology);

  // Nvidia PCI vender ID 
  // https://devicehunt.com/view/type/pci/vendor/10DE/device/20B0
  unsigned short nv_vender_id = 0x10DE;  // 4318
  hwloc_obj_t gpu_obj = nullptr;
  while ((gpu_obj = hwloc_get_next_pcidev(topology, gpu_obj)) != nullptr) {
    if (gpu_obj->attr->pcidev.vendor_id == nv_vender_id) {
      printf("     %hu, %hu, %hu,\n", 
            gpu_obj->attr->pcidev.domain, gpu_obj->attr->pcidev.bus, gpu_obj->attr->pcidev.dev);
    }
  }

  return 0;
}